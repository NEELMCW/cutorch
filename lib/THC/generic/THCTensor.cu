
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensor.cu"
#else

hipTextureObject_t THCTensor_(getTextureObject)(THCState *state, THCTensor *self)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  hipTextureObject_t texObj;
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = THCTensor_(data)(state, self);
  resDesc.res.linear.sizeInBytes = THCTensor_(nElement)(state, self) * 4;
  resDesc.res.linear.desc = hipCreateChannelDesc(32, 0, 0, 0,
                                                  hipChannelFormatKindFloat);
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess) {
    if (THCTensor_(nElement)(state, self) > 2>>27)
      THError("Failed to create texture object, "
              "nElement:%ld exceeds 27-bit addressing required for tex1Dfetch. Cuda Error: %s",
              THCTensor_(nElement)(state, self), hipGetErrorString(errcode));
    else
      THError("Failed to create texture object: %s", hipGetErrorString(errcode));
  }
  return texObj;
}

THC_API int THCTensor_(getDevice)(THCState* state, const THCTensor* tensor) {
  if (!tensor->storage) return -1;
  return THCStorage_(getDevice)(state, tensor->storage);
}

#endif
